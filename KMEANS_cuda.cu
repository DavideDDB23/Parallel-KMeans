/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <hip/hip_runtime.h>


#define MAXLINE 2000
#define MAXCAD 200

//Macros
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL(a) { \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

/* 
Function showFileError: It displays the corresponding error during file reading.
*/
void showFileError(int error, char* filename)
{
	printf("Error\n");
	switch (error)
	{
		case -1:
			fprintf(stderr,"\tFile %s has too many columns.\n", filename);
			fprintf(stderr,"\tThe maximum number of columns has been exceeded. MAXLINE: %d.\n", MAXLINE);
			break;
		case -2:
			fprintf(stderr,"Error reading file: %s.\n", filename);
			break;
		case -3:
			fprintf(stderr,"Error writing file: %s.\n", filename);
			break;
	}
	fflush(stderr);	
}

/* 
Function readInput: It reads the file to determine the number of rows and columns.
*/
int readInput(char* filename, int *lines, int *samples)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int contlines, contsamples = 0;
    
    contlines = 0;

    if ((fp=fopen(filename,"r"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL) 
		{
			if (strchr(line, '\n') == NULL)
			{
				return -1;
			}
            contlines++;       
            ptr = strtok(line, delim);
            contsamples = 0;
            while(ptr != NULL)
            {
            	contsamples++;
				ptr = strtok(NULL, delim);
	    	}	    
        }
        fclose(fp);
        *lines = contlines;
        *samples = contsamples;  
        return 0;
    }
    else
	{
    	return -2;
	}
}

/* 
Function readInput2: It loads data from file.
*/
int readInput2(char* filename, float* data)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int i = 0;
    
    if ((fp=fopen(filename,"rt"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL)
        {         
            ptr = strtok(line, delim);
            while(ptr != NULL)
            {
            	data[i] = atof(ptr);
            	i++;
				ptr = strtok(NULL, delim);
	   		}
	    }
        fclose(fp);
        return 0;
    }
    else
	{
    	return -2; //No file found
	}
}

/* 
Function writeResult: It writes in the output file the cluster of each sample (point).
*/
int writeResult(int *classMap, int lines, const char* filename)
{	
    FILE *fp;
    
    if ((fp=fopen(filename,"wt"))!=NULL)
    {
        for(int i=0; i<lines; i++)
        {
        	fprintf(fp,"%d\n",classMap[i]);
        }
        fclose(fp);  
   
        return 0;
    }
    else
	{
    	return -3; //No file found
	}
}

/*
Function initCentroids: This function copies the values of the initial centroids, using their 
position in the input data structure as a reference map.
*/
void initCentroids(const float *data, float* centroids, int* centroidPos, int samples, int K)
{
	int i;
	int idx;
	for(i = 0; i < K; i++) {
		idx = centroidPos[i];
		memcpy(&centroids[i * samples], &data[idx * samples], (samples * sizeof(float)));
	}
}

//-------------------------------------------------------------
// CUDA Kernels and Device Functions
//-------------------------------------------------------------

// Implementation of a custom atomicMax operation for floats.
__device__ inline float custom_atomic_max(float *value_address, float val)
{
    int *address_as_int = (int *)value_address;
    int old = *address_as_int, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_int, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

/*
 * step_1_kernel:
 *   - Each thread computes the nearest centroid for one data point.
 *   - Uses dynamic shared memory to copy the centroids and to store block-level
 *     accumulators (sums and counts) for centroid updates.
 *   - Updates the global assignment array (classMap) and counts the number of changes.
 *
 * Dynamic shared memory layout:
 *   [sharedCentroids | blockSums | blockCounts]
 *     - sharedCentroids: K * samples floats (a copy of the centroids)
 *     - blockSums: K * samples floats (partial sums for each centroid)
 *     - blockCounts: K ints (number of points assigned per centroid)
 */
__global__ void step_1_kernel( float *  data,
                               float *  centroids,
                              int *globalCounts,
                              float *globalSums,
                              int *classMap,
                              int *changes_return,
                              int lines,
                              int samples,
                              int K)
{
    // Flatten thread index (we assume grid.y == 1)
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int blockSize = blockDim.x * blockDim.y;
    int idx = blockIdx.x * blockSize + tid;

    // Allocate dynamic shared memory:
    extern __shared__ char sharedBuffer[];
    float *sharedCentroids = (float *)sharedBuffer;                  // K * samples floats.
    float *blockSums = sharedCentroids + K * samples;                  // K * samples floats.
    int   *blockCounts = (int *)(blockSums + K * samples);             // K ints.

    // A block-level shared variable to count how many assignments changed in this block.
    __shared__ int blockChanges;

    // Copy centroids into shared memory.
    for (int i = tid; i < K * samples; i += blockSize)
    {
        sharedCentroids[i] = centroids[i];
    }
    // Initialize blockSums and blockCounts to zero.
    for (int i = tid; i < K * samples; i += blockSize)
    {
        blockSums[i] = 0.0f;
    }
    for (int i = tid; i < K; i += blockSize)
    {
        blockCounts[i] = 0;
    }
    if (tid == 0)
    {
        blockChanges = 0;
    }
    __syncthreads();

    if (idx < lines)
    {
        const float *point = &data[idx * samples];
        int best_cluster = 0;
        float best_dist = FLT_MAX;
        // Loop over centroids and compute squared Euclidean distance.
        for (int c = 0; c < K; c++)
        {
            float dist = 0.0f;
            // Unroll the inner loop if samples is small.
#pragma unroll
            for (int j = 0; j < samples; j++)
            {
                float diff = point[j] - sharedCentroids[c * samples + j];
                dist += diff * diff;
            }
            if (dist < best_dist)
            {
                best_dist = dist;
                best_cluster = c;
            }
        }

        // Note: classMap stores 1-indexed assignments.
        int old_cluster = classMap[idx] - 1;
        if (old_cluster != best_cluster)
        {
            atomicAdd(&blockChanges, 1);
        }
        classMap[idx] = best_cluster + 1;

        // Update block-level accumulators.
        atomicAdd(&blockCounts[best_cluster], 1);
        for (int j = 0; j < samples; j++)
        {
            atomicAdd(&blockSums[best_cluster * samples + j], point[j]);
        }
    }
    __syncthreads();

    // One thread (tid==0) per block updates the global accumulators.
    if (tid == 0)
    {
        atomicAdd(changes_return, blockChanges);
        for (int c = 0; c < K; c++)
        {
            atomicAdd(&globalCounts[c], blockCounts[c]);
            for (int j = 0; j < samples; j++)
            {
                atomicAdd(&globalSums[c * samples + j], blockSums[c * samples + j]);
            }
        }
    }
}

/*
 * step_2_kernel:
 *   - Each thread processes one cluster (centroid).
 *   - The new centroid is computed by averaging the sums in globalSums (from step 1) divided by the count.
 *   - The squared Euclidean distance between the old and new centroid is computed.
 *   - A custom atomic max is used to update the global maximum centroid movement.
 */
__global__ void step_2_kernel(float *globalSums,
                              float *centroids,
                              int *globalCounts,
                              float *maxDistance,
                              int samples,
                              int K)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c < K)
    {
        float dist = 0.0f;
        if (globalCounts[c] > 0)
        {
            for (int j = 0; j < samples; j++)
            {
                float newVal = globalSums[c * samples + j] / (float)globalCounts[c];
                float diff = centroids[c * samples + j] - newVal;
                dist += diff * diff;
                centroids[c * samples + j] = newVal;
            }
        }
        custom_atomic_max(maxDistance, dist);
    }
}

int main(int argc, char *argv[])
{
    // START CLOCK (for overall timing)
    clock_t start, end;
    start = clock();

    // PARAMETERS:
    // argv[1]: Input data file
    // argv[2]: Number of clusters
    // argv[3]: Maximum number of iterations
    // argv[4]: Percentage of points that must change to continue
    // argv[5]: Threshold (centroid movement) for convergence
    // argv[6]: Output file (each line: cluster assignment, 1-indexed)
    if (argc != 7)
    {
        fprintf(stderr, "EXECUTION ERROR K-MEANS: Parameters are not correct.\n");
        fprintf(stderr, "./KMEANS [Input Filename] [Number of clusters] [Number of iterations] [Percentage of changes] [Threshold] [Output data file]\n");
        fflush(stderr);
        exit(-1);
    }

    int lines = 0, samples = 0;
    int error = readInput(argv[1], &lines, &samples);
    if (error != 0)
    {
        showFileError(error, argv[1]);
        exit(error);
    }

    float *data = (float *)calloc(lines * samples, sizeof(float));
    if (data == NULL)
    {
        fprintf(stderr, "Memory allocation error.\n");
        exit(-4);
    }
    error = readInput2(argv[1], data);
    if (error != 0)
    {
        showFileError(error, argv[1]);
        exit(error);
    }

    // Parameters from command line.
    int K = atoi(argv[2]);
    int maxIterations = atoi(argv[3]);
    int minChanges = (int)(lines * atof(argv[4]) / 100.0);
    float maxThreshold = atof(argv[5]);

    int *centroidPos = (int *)calloc(K, sizeof(int));
    float *centroids = (float *)calloc(K * samples, sizeof(float));
    int *classMap = (int *)calloc(lines, sizeof(int));
    if (centroidPos == NULL || centroids == NULL || classMap == NULL)
    {
        fprintf(stderr, "Memory allocation error.\n");
        exit(-4);
    }

    // Initialize centroids randomly.
    srand(0);
    for (int i = 0; i < K; i++)
    {
        centroidPos[i] = rand() % lines;
    }
    initCentroids(data, centroids, centroidPos, samples, K);

    printf("\n    Input properties:");
    printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", argv[1], lines, samples);
    printf("\tNumber of clusters: %d\n", K);
    printf("\tMaximum number of iterations: %d\n", maxIterations);
    printf("\tMinimum number of changes: %d [%g%% of %d points]\n", minChanges, atof(argv[4]), lines);
    printf("\tMaximum centroid precision: %f\n", maxThreshold);

    // Retrieve and display CUDA device properties.
    hipDeviceProp_t cuda_prop;
    CHECK_CUDA_CALL(hipGetDeviceProperties(&cuda_prop, 0));
    printf("\n    Device: %s\n", cuda_prop.name);
    printf("\tCompute Capability: %d.%d\n", cuda_prop.major, cuda_prop.minor);
    printf("\tMax threads / block: %d\n", cuda_prop.maxThreadsPerBlock);
    printf("\tMax threads / SM: %d\n", cuda_prop.maxThreadsPerMultiProcessor);
    printf("\tMax shared memory per SM: %zuB\n", cuda_prop.sharedMemPerMultiprocessor);
    printf("\tNumber of SMs: %d\n", cuda_prop.multiProcessorCount);

    CHECK_CUDA_CALL(hipSetDevice(0));
    CHECK_CUDA_CALL(hipDeviceSynchronize());

    // Allocate host memory for some intermediate results.
    char *output_msg = (char *)calloc(100000, sizeof(char));
    int it = 0;
    int changes = 0;
    float maxDist = 0.0f;
    // Precompute threshold squared (avoid pow() in loop).
    float thresholdSq = maxThreshold * maxThreshold;

    int *pointsPerClass = (int *)malloc(K * sizeof(int));
    float *auxCentroids = (float *)malloc(K * samples * sizeof(float));
    if (pointsPerClass == NULL || auxCentroids == NULL)
    {
        fprintf(stderr, "Memory allocation error.\n");
        exit(-4);
    }

    // Calculate dynamic shared memory needed for step_1_kernel.
    int sharedMemSize = 2 * K * samples * sizeof(float) + K * sizeof(int);

    // Determine grid dimensions for step_1_kernel.
    // We use a 2D block configuration of 32x32 threads (1024 threads per block).
    dim3 gen_block(32, 32);
    int threadsPerBlock = gen_block.x * gen_block.y;
    int numBlocks = (lines + threadsPerBlock - 1) / threadsPerBlock;
    dim3 dyn_grid_pts(numBlocks, 1);

    // Grid configuration for step_2_kernel: each thread processes one cluster.
    int threadsPerBlock2 = 256;
    int blocksForClusters = (K + threadsPerBlock2 - 1) / threadsPerBlock2;

    // ------------------------------------------------------------
    // GPU Memory Allocation and Data Transfer
    // ------------------------------------------------------------
    float *gpu_data;
    float *gpu_centroids;
    int *gpu_class_map;
    float *gpu_aux_centroids;
    int *gpu_points_per_class;
    int *gpu_changes;
    float *gpu_max_distance;

    int data_size = lines * samples * sizeof(float);
    int centroids_size = K * samples * sizeof(float);

    CHECK_CUDA_CALL(hipMalloc((void **)&gpu_centroids, centroids_size));
    CHECK_CUDA_CALL(hipMemcpy(gpu_centroids, centroids, centroids_size, hipMemcpyHostToDevice));

    CHECK_CUDA_CALL(hipMalloc((void **)&gpu_data, data_size));
    CHECK_CUDA_CALL(hipMemcpy(gpu_data, data, data_size, hipMemcpyHostToDevice));

    // Allocate device memory for the cluster assignments.
    CHECK_CUDA_CALL(hipMalloc((void **)&gpu_class_map, lines * sizeof(int)));
    CHECK_CUDA_CALL(hipMemset(gpu_class_map, 0, lines * sizeof(int)));

    // Allocate device memory for auxiliary centroids (to accumulate sums).
    CHECK_CUDA_CALL(hipMalloc((void **)&gpu_aux_centroids, centroids_size));
    CHECK_CUDA_CALL(hipMemset(gpu_aux_centroids, 0, centroids_size));

    // Allocate device memory for points per cluster.
    CHECK_CUDA_CALL(hipMalloc((void **)&gpu_points_per_class, K * sizeof(int)));
    CHECK_CUDA_CALL(hipMemset(gpu_points_per_class, 0, K * sizeof(int)));

    // Allocate device memory for the change counter.
    CHECK_CUDA_CALL(hipMalloc((void **)&gpu_changes, sizeof(int)));
    // Allocate device memory for tracking maximum centroid movement.
    CHECK_CUDA_CALL(hipMalloc((void **)&gpu_max_distance, sizeof(float)));

    end = clock();
    printf("\nMemory allocation and initialization: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
    fflush(stdout);

    // START CUDA computation clock
    start = clock();

    // ------------------------------------------------------------
    // Main k-Means Iterative Loop (on the GPU)
    // ------------------------------------------------------------
    do
    {
        it++;
        // Reset accumulators on the GPU.
        CHECK_CUDA_CALL(hipMemset(gpu_changes, 0, sizeof(int)));
        CHECK_CUDA_CALL(hipMemcpy(gpu_max_distance, 0, sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA_CALL(hipMemset(gpu_aux_centroids, 0, centroids_size));
        CHECK_CUDA_CALL(hipMemset(gpu_points_per_class, 0, K * sizeof(int)));

        // Launch step_1_kernel: each thread processes one data point.
        step_1_kernel<<<dyn_grid_pts, gen_block, sharedMemSize>>>(gpu_data,
                                                                    gpu_centroids,
                                                                    gpu_points_per_class,
                                                                    gpu_aux_centroids,
                                                                    gpu_class_map,
                                                                    gpu_changes,
                                                                    lines,
                                                                    samples,
                                                                    K);
        CHECK_CUDA_LAST();
        CHECK_CUDA_CALL(hipDeviceSynchronize());

        // Retrieve the number of changed assignments.
        CHECK_CUDA_CALL(hipMemcpy(&changes, gpu_changes, sizeof(int), hipMemcpyDeviceToHost));

        // Launch step_2_kernel: update each centroid and compute its movement.
        step_2_kernel<<<blocksForClusters, threadsPerBlock2>>>(gpu_aux_centroids,
                                                               gpu_centroids,
                                                               gpu_points_per_class,
                                                               gpu_max_distance,
                                                               samples,
                                                               K);
        CHECK_CUDA_LAST();
        CHECK_CUDA_CALL(hipDeviceSynchronize());

        // Copy the maximum centroid movement back to the host.
        CHECK_CUDA_CALL(hipMemcpy(&maxDist, gpu_max_distance, sizeof(float), hipMemcpyDeviceToHost));

    } while ((changes > minChanges) && (it < maxIterations) && (maxDist > thresholdSq));

    printf("%s", output_msg);
    CHECK_CUDA_CALL(hipDeviceSynchronize());

    end = clock();
    printf("\nComputation: %f seconds", (double)(end - start) / CLOCKS_PER_SEC);
    fflush(stdout);
    start = clock();

    if (changes <= minChanges)
    {
        printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, minChanges);
    }
    else if (it >= maxIterations)
    {
        printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, maxIterations);
    }
    else
    {
        printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", maxDist, maxThreshold);
    }

    // Copy final cluster assignments from the GPU.
    CHECK_CUDA_CALL(hipMemcpy(classMap, gpu_class_map, lines * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA_CALL(hipDeviceSynchronize());

    // Write the results to the output file.
    error = writeResult(classMap, lines, argv[6]);
    if (error != 0)
    {
        showFileError(error, argv[6]);
        exit(error);
    }

    // Free host and device memory.
    free(data);
    free(classMap);
    free(centroidPos);
    free(centroids);
    free(pointsPerClass);
    free(auxCentroids);

    hipFree(gpu_data);
    hipFree(gpu_centroids);
    hipFree(gpu_aux_centroids);
    hipFree(gpu_changes);
    hipFree(gpu_class_map);
    hipFree(gpu_max_distance);
    hipFree(gpu_points_per_class);

    end = clock();
    printf("\n\nMemory deallocation: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
    fflush(stdout);
    return 0;
}